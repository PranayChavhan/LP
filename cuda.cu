
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;
using namespace std::chrono;

// ---------------------- VECTOR ADDITION --------------------------
__global__ void vectorAddCUDA(int *a, int *b, int *c, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

void vectorAddCPU(int *a, int *b, int *c, int n)
{
    for (int i = 0; i < n; i++)
        c[i] = a[i] + b[i];
    // for (int i = 0; i < n; i++)
    //     cout << c[i] << " ";
    // cout << endl;
}

// ---------------------- MATRIX MULTIPLICATION ---------------------
__global__ void matrixMulCUDA(int *a, int *b, int *c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        int sum = 0;
        for (int k = 0; k < N; k++)
            sum += a[row * N + k] * b[k * N + col];
        c[row * N + col] = sum;
    }
}

void matrixMulCPU(int *a, int *b, int *c, int N)
{
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
        {
            int sum = 0;
            for (int k = 0; k < N; k++)
                sum += a[i * N + k] * b[k * N + j];
            c[i * N + j] = sum;
        }

    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         cout << c[i * N + j] << " ";
    //     }
    //     cout << endl;
    // }
}

// ------------------------ MAIN FUNCTION ---------------------------
int main()
{
    const int vecSize = 1 << 24; // ~16 million
    const int matrixSize = 1024; // 1024x1024 matrix

    // ------------------ Vector Addition ------------------
    int *h_a = new int[vecSize];
    int *h_b = new int[vecSize];
    int *h_c_cpu = new int[vecSize];
    int *h_c_gpu = new int[vecSize];

    for (int i = 0; i < vecSize; ++i)
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    auto start = high_resolution_clock::now();
    vectorAddCPU(h_a, h_b, h_c_cpu, vecSize);
    auto end = high_resolution_clock::now();
    cout << "[Vector Addition - CPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, vecSize * sizeof(int));
    hipMalloc(&d_b, vecSize * sizeof(int));
    hipMalloc(&d_c, vecSize * sizeof(int));

    hipMemcpy(d_a, h_a, vecSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vecSize * sizeof(int), hipMemcpyHostToDevice);

    start = high_resolution_clock::now();
    vectorAddCUDA<<<(vecSize + 255) / 256, 256>>>(d_a, d_b, d_c, vecSize);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();

    hipMemcpy(h_c_gpu, d_c, vecSize * sizeof(int), hipMemcpyDeviceToHost);
    cout << "[Vector Addition - GPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n\n";

    // ------------------ Matrix Multiplication ------------------
    int *matA = new int[matrixSize * matrixSize];
    int *matB = new int[matrixSize * matrixSize];
    int *matC_cpu = new int[matrixSize * matrixSize];
    int *matC_gpu = new int[matrixSize * matrixSize];

    for (int i = 0; i < matrixSize * matrixSize; ++i)
    {
        matA[i] = rand() % 100;
        matB[i] = rand() % 100;
    }

    start = high_resolution_clock::now();
    matrixMulCPU(matA, matB, matC_cpu, matrixSize);
    end = high_resolution_clock::now();
    cout << "[Matrix Multiplication - CPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    int *d_matA, *d_matB, *d_matC;
    size_t bytes = matrixSize * matrixSize * sizeof(int);
    hipMalloc(&d_matA, bytes);
    hipMalloc(&d_matB, bytes);
    hipMalloc(&d_matC, bytes);

    hipMemcpy(d_matA, matA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((matrixSize + 15) / 16, (matrixSize + 15) / 16);

    start = high_resolution_clock::now();
    matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_matA, d_matB, d_matC, matrixSize);
    hipDeviceSynchronize();
    end = high_resolution_clock::now();
    hipMemcpy(matC_gpu, d_matC, bytes, hipMemcpyDeviceToHost);

    cout << "[Matrix Multiplication - GPU] Time: " << duration_cast<milliseconds>(end - start).count() << " ms\n";

    // Cleanup
    delete[] h_a;
    delete[] h_b;
    delete[] h_c_cpu;
    delete[] h_c_gpu;
    delete[] matA;
    delete[] matB;
    delete[] matC_cpu;
    delete[] matC_gpu;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    return 0;
}



// Commands to run
// nvcc Assignment4.cu -o output
// ./output



// This CUDA C++ program benchmarks parallel (GPU) vs. sequential (CPU) 
// computation for two fundamental tasks: vector addition and matrix multiplication. 
// It first allocates large arrays—~16 million elements for vectors and 1024×1024 matrices—using dynamic memory. 
// In the vector addition section, it performs element-wise addition of vectors a and b to get c, 
// first using a CPU function and then a CUDA kernel. The GPU kernel calculates the global thread 
// index and performs the addition only if the index is within bounds. CUDA memory allocation (cudaMalloc) 
// and data transfers (cudaMemcpy) are used to move data between host (CPU) and device (GPU). For matrix multiplication, 
// it computes each cell of the output matrix c as the dot product of the corresponding row and column 
// from input matrices a and b. The GPU kernel leverages 2D thread and block indexing to handle rows 
// and columns, significantly accelerating the operation by exploiting massive parallelism. 
// High-resolution timers measure execution times, highlighting how GPU computation—despite overhead 
// from memory transfers—can outperform the CPU, especially for compute-heavy tasks like matrix multiplication. 
// Finally, all dynamically allocated memory is cleaned up to prevent leaks. The code effectively demonstrates 
// how CUDA can accelerate numerical tasks by parallelizing them across hundreds or thousands of threads.